#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <memory>
#include <string>

#include <hip/hip_runtime.h>

hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    using namespace std::literals::string_literals;
    throw std::runtime_error("CUDA Runtime Error : "s + hipGetErrorString(result));
    //assert(result == hipSuccess);
  }
  return result;
}

int main(int argc, char **argv) 
{

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) 
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

        // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) 
    {
        printf("There are no available device(s) that support CUDA\n");
    } 
    else 
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

}